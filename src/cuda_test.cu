#include <iostream>
#include <hip/hip_runtime.h>

// Simple CUDA kernel that does nothing but prove CUDA works
__global__ void helloKernel() {
    // Empty kernel, just to test CUDA compilation and execution
}

// Function declaration for external linkage
extern "C" bool testCuda();

// Function to test basic CUDA functionality
bool testCuda() {
    // Print CUDA device properties
    int deviceCount = 0;
    hipError_t error = hipGetDeviceCount(&deviceCount);
    
    if (error != hipSuccess) {
        std::cerr << "Error getting CUDA device count: " 
                  << hipGetErrorString(error) << std::endl;
        return false;
    }
    
    std::cout << "Found " << deviceCount << " CUDA device(s)" << std::endl;
    
    // Get properties for each device
    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t prop;
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
            std::cout << "Device " << i << ": " << prop.name << std::endl;
            std::cout << "  Compute capability: " << prop.major << "." << prop.minor << std::endl;
            std::cout << "  Total global memory: " << prop.totalGlobalMem / (1024 * 1024) << " MB" << std::endl;
            std::cout << "  Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
        }
    }
    
    // Launch an empty kernel just to verify CUDA execution works
    helloKernel<<<1, 1>>>();
    
    // Check for kernel launch errors
    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(kernelError) << std::endl;
        return false;
    }
    
    // Wait for GPU to finish
    if (hipDeviceSynchronize() != hipSuccess) {
        std::cerr << "CUDA synchronize failed: " << hipGetErrorString(hipGetLastError()) << std::endl;
        return false;
    }
    
    std::cout << "CUDA kernel executed successfully!" << std::endl;
    return true;
}